#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>
#include <chrono>
#include <limits>

static double inf = std::numeric_limits<double>::max();

using namespace std::chrono;

int main() {
    thrust::host_vector<double> host;
    std::string line;
    double s;
    high_resolution_clock::time_point start, end;
    
    while (!std::cin.eof()) {
        std::cin >> s;
        host.push_back(s);
    }

    /* na linha abaixo os dados são copiados
       para GPU */
    start = high_resolution_clock::now();
    thrust::device_vector<double> dev(host);    
    end = high_resolution_clock::now();
    double d = duration_cast<milliseconds>(end - start).count();

    std::cerr << "ASSIGN AND COPY TO GPU:" << d << " ms" << std::endl;
    
    // printf("Device vector: ");
    // for (auto i = dev.begin(); i != dev.end(); i++) {
    //     std::cout << *i << " "; // este acesso é lento! -- GPU
    // }

    start = high_resolution_clock::now();
    double sum = thrust::reduce(dev.begin(), dev.end(), 0, thrust::plus<double>());
    double avg = sum/host.size();
    double sum_last_year = thrust::reduce(dev.end() - 365, dev.end(), 0, thrust::plus<double>());
    double avg_last_year = sum_last_year/365;
    double max = thrust::reduce(dev.begin(), dev.end(), 0, thrust::maximum<double>());
    double min = thrust::reduce(dev.begin(), dev.end(), inf, thrust::minimum<double>());
    double max_ly = thrust::reduce(dev.end() - 365, dev.end(), 0, thrust::maximum<double>());
    double min_ly = thrust::reduce(dev.end() - 365, dev.end(), inf, thrust::minimum<double>());
    end = high_resolution_clock::now();
    d = duration_cast<nanoseconds>(end - start).count();
    std::cerr << "OPERATIONS IN GPU:" << d << " ns" << std::endl;

    std::cout << "max: " << max << std::endl;
    std::cout << "min: " << min << std::endl;
    std::cout << "max_ly: " << max_ly << std::endl;
    std::cout << "min_ly: " << min_ly << std::endl;
    std::cout << "avg: " << avg << std::endl;
    std::cout << "avg_ly: " << avg_last_year << std::endl;
    std::cout << "sum: " << sum << std::endl;

    start = high_resolution_clock::now();
    sum = thrust::reduce(thrust::host, host.begin(), host.end(), 0, thrust::plus<double>());
    avg = sum/host.size();
    sum_last_year = thrust::reduce(thrust::host, host.end() - 365, host.end(), 0, thrust::plus<double>());
    avg_last_year = sum_last_year/365;
    max = thrust::reduce(thrust::host, host.begin(), host.end(), 0, thrust::maximum<double>());
    min = thrust::reduce(thrust::host, host.begin(), host.end(), inf, thrust::minimum<double>());
    max_ly = thrust::reduce(thrust::host, host.end() - 365, host.end(), 0, thrust::maximum<double>());
    min_ly = thrust::reduce(thrust::host, host.end() - 365, host.end(), inf, thrust::minimum<double>());
    end = high_resolution_clock::now();
    d = duration_cast<nanoseconds>(end - start).count();
    std::cerr << "OPERATIONS IN CPU:" << d << " ns" << std::endl;

    std::cout << "max: " << max << std::endl;
    std::cout << "min: " << min << std::endl;
    std::cout << "max_ly: " << max_ly << std::endl;
    std::cout << "min_ly: " << min_ly << std::endl;
    std::cout << "avg: " << avg << std::endl;
    std::cout << "avg_ly: " << avg_last_year << std::endl;
    std::cout << "sum: " << sum << std::endl;

}
