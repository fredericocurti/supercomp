#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>
#include <chrono>
#include <limits>

static double inf = std::numeric_limits<double>::max();

using namespace std::chrono;

int main() {
    double s;
    thrust::host_vector<double> host_AAPL;
    thrust::host_vector<double> host_MSFT;
    std::string line;
    high_resolution_clock::time_point start, end;
    
    while (!std::cin.eof()) {
        std::cin >> s;
        host_AAPL.push_back(s);
        std::cin >> s;
        host_MSFT.push_back(s);
    }

    /* na linha abaixo os dados são copiados
       para GPU */
    start = high_resolution_clock::now();
    thrust::device_vector<double> dev_AAPL(host_AAPL);    
    thrust::device_vector<double> dev_MSFT(host_MSFT);
    thrust::device_vector<double> dev(host_AAPL.size());
    end = high_resolution_clock::now();
    double d = duration_cast<milliseconds>(end - start).count();
    std::cerr << "ASSIGN AND COPY TO GPU:" << d << " ms" << std::endl;
    
    // printf("Device vector: ");
    // for (auto i = dev.begin(); i != dev.end(); i++) {
    //     std::cout << *i << " "; // este acesso é lento! -- GPU
    // }

    start = high_resolution_clock::now();
    thrust::transform(dev_AAPL.begin(), dev_AAPL.end(), dev_MSFT.begin(), dev.begin(), thrust::minus<double>());
    double avg = thrust::reduce(dev.begin(), dev.end(), 0, thrust::plus<double>());
    avg = avg/host_AAPL.size();

    std::cout << avg << std::endl;

    return 0;
}
