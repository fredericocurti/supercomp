#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <iostream>
#include <chrono>
#include <omp.h>

static long STEPS = 1000000000;
static int THREADS_PER_BLOCK = 1024;
static double STEP = 1.0 / (double) STEPS;

__global__ void pi_cuda(double *a, double step) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    double x = (i + 0.5) * step;
    a[i] = 4.0 / (1.0 + x * x);
}

double pi_omp_parallel_for() {
    double sum;
    double x = 0;
    #pragma omp parallel for reduction(+:sum) firstprivate(x)
    for (int i = 0; i < STEPS; i++) {
        x = (i + 0.5) * STEP;
        sum = sum + 4.0 / (1.0 + x * x);
    }

    return STEP * sum;
}

int main(int argc, char **argv) {
    thrust::device_vector<double> v(STEPS);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, NULL);
    pi_cuda<<<ceil((double) STEPS/THREADS_PER_BLOCK), THREADS_PER_BLOCK>>>(thrust::raw_pointer_cast(v.data()), STEP);
    double sum = thrust::reduce(v.begin(), v.end(), 0.0, thrust::plus<double>());
    double pi = sum * STEP;

    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);

    // Compute and print the performance 
    printf("Time= %2.5f\n",msecTotal);
    std::cout << pi << std::endl;

    auto start_time = std::chrono::high_resolution_clock::now();
    pi = pi_omp_parallel_for();
    auto end_time = std::chrono::high_resolution_clock::now();
    auto runtime = std::chrono::duration_cast<std::chrono::milliseconds> (end_time - start_time);
    std::cout << "Time(omp)" << runtime.count() << std::endl;
    std::cout << pi << std::endl;
}
