#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <iostream>
#include <chrono>
#include <omp.h>
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

static double TRIALS = 100000000;
static int THREADS_PER_BLOCK = 1024;
// static double STEP = 1.0 / (double) STEPS;

__global__ void pi_cuda_mc(double *v, double r, int sz = 0) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    double x, y, test;
    hiprandState st;
    hiprand_init(0, i, 0, &st);

    if (sz == 0) {
            // números entre -1 e 1
        x = (double) (2 * hiprand_uniform(&st) - 1);
        y = (double) (2 * hiprand_uniform(&st) - 1);

        test = x * x + y * y;
        v[i] = (test <= r * r) ? 1 : 0;
        return;
    } else {
        long start = i * sz;
        long end = (i+1) * sz;
        double sum = 0.0;
        for (int k = start; k < end; k++) {
            x = (double) (2 * hiprand_uniform(&st) - 1);
            y = (double) (2 * hiprand_uniform(&st) - 1);
            test = x * x + y * y;
            if (test <= r * r) {
                sum++;        
            }
        }
        v[i] = sum;
    }
    
}

// double pi_omp_parallel_for() {
//     double sum;
//     double x = 0;
//     #pragma omp parallel for reduction(+:sum) firstprivate(x)
//     for (int i = 0; i < STEPS; i++) {
//         x = (i + 0.5) * STEP;
//         sum = sum + 4.0 / (1.0 + x * x);
//     }

//     return STEP * sum;
// }

int ingenuo() {
    thrust::device_vector<double> v(TRIALS);
    double r = 1.0; // radius of circle. Side of squrare is 2*r

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, NULL);
    pi_cuda_mc<<<ceil((double) TRIALS/THREADS_PER_BLOCK), THREADS_PER_BLOCK>>>(
        thrust::raw_pointer_cast(v.data()),
        r
    );

    double Ncirc = thrust::reduce(v.begin(), v.end(), 0.0, thrust::plus<double>());
    double pi = 4 * (Ncirc / TRIALS);

    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);

    std::cout << "trials:" << TRIALS << "pi is" << pi << std::endl;
    std::cout << msecTotal << " ms" << std::endl;

    // auto start_time = std::chrono::high_resolution_clock::now();
    // pi = pi_omp_parallel_for();
    // auto end_time = std::chrono::high_resolution_clock::now();
    // auto runtime = std::chrono::duration_cast<std::chrono::milliseconds> (end_time - start_time);
    // std::cout << "Time(omp)" << runtime.count() << std::endl;
    // std::cout << pi << std::endl;
    return 0;
}

int esperto() {
    int els_per_thread = 8192;
    thrust::device_vector<double> v(TRIALS/els_per_thread);
    double r = 1.0; // radius of circle. Side of squrare is 2*r
        
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, NULL);
    pi_cuda_mc<<<ceil((double) TRIALS/(THREADS_PER_BLOCK * els_per_thread)), THREADS_PER_BLOCK>>>(
        thrust::raw_pointer_cast(v.data()),
        r,
        els_per_thread
    );

    double Ncirc = thrust::reduce(v.begin(), v.end(), 0.0, thrust::plus<double>());
    double pi = 4 * (Ncirc / TRIALS);

    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);

    std::cout << "trials: " << TRIALS << " pi is " << pi << std::endl;
    std::cout << msecTotal << " ms" << std::endl;

    // auto start_time = std::chrono::high_resolution_clock::now();
    // pi = pi_omp_parallel_for();
    // auto end_time = std::chrono::high_resolution_clock::now();
    // auto runtime = std::chrono::duration_cast<std::chrono::milliseconds> (end_time - start_time);
    // std::cout << "Time(omp)" << runtime.count() << std::endl;
    // std::cout << pi << std::endl;
    return 0;
}

int main(int argc, char **argv) {
    esperto();
}