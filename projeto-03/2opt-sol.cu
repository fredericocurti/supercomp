#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <iomanip>
#include <cmath>
#include <chrono>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

typedef struct {
    double x;
    double y;
} point;

__device__ double dist(point p1, point p2) {
    return sqrt(pow(p1.x - p2.x, 2) + pow(p1.y - p2.y, 2));
}

__device__ double cost(int *d_solutions, double *d_distances, int i, int N) {
    double c = 0;
    for (int j = 1; j < N; j++) {
        c += d_distances[d_solutions[i * N + (j - 1)] * N + d_solutions[i * N + j]];
    }
    c += d_distances[d_solutions[i * N] * N + d_solutions[i * N + (N - 1)]];
    return c;
}

__device__ void print_vec_int(int *v, int N) {
    for (int i = 0; i < N; i++) {
        printf("%d ", v[i]);
    }
    printf("\n");
}

__device__ void swap(int *a, int *b) {
    int temp = *a;
    *a = *b;
    *b = temp;
    return;
}

__global__ void distKernel(point *points, double *d_distances, int N) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N || j >= N) return;
    d_distances[i * N + j] = dist(points[i], points[j]);
}

__global__ void solKernel(double *d_distances, double *d_costs, int *d_solutions, int N) {
    hiprandState st;
    double swap_cost = 0; // current cost for 2opt
    double best_cost = 0; // solution cost (smallest)
    int ri; // random index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    hiprand_init(0, i, 0, &st);

    // Fill solution with sequential possible N's
    for (int j = 0; j < N; j++) {
        d_solutions[i * N + j] = j;
    }

    // Swap from random indexes to create a new possible solution
    for (int j = 1; j < N; j++) {
        ri = (int) ((N - j) * hiprand_uniform(&st) + j); // (max - min) * hiprand + min
        swap(&d_solutions[i * N + j], &d_solutions[i * N + ri]);
        best_cost += d_distances[d_solutions[i * N + (j - 1)] * N + d_solutions[i * N + j]];
    }

    // Add cost between first and last
    best_cost += d_distances[d_solutions[i * N] * N + d_solutions[i * N + (N - 1)]];
    d_costs[i] = best_cost;

    // Swap again for 2opt
    for (int j = 1; j < N; j++) {
        for (int k = j + 1; k < N; k++) {
            swap(&d_solutions[i * N + j], &d_solutions[i * N + k]);
            swap_cost = cost(d_solutions, d_distances, i, N);
            if (swap_cost > best_cost) { // undo
                swap(&d_solutions[i * N + k], &d_solutions[i * N + j]);
            } else { // keep the solution
                best_cost = swap_cost;
            }
        }
    }
    d_costs[i] = best_cost;
}

int main(int argc, char *argv[]) {
    // Variable declaration
    int N, min_cost_pos;
    int SOLUTIONS = atoi(argv[1]);
    double min_cost;
    float msecTotal = 0.0f;
    point p;
    std::cin >> N;
    thrust::host_vector<point> h_points(N);
    thrust::device_vector<double> d_distances(N * N);
    thrust::device_vector<double> d_costs(SOLUTIONS);
    thrust::device_vector<int> d_solutions(SOLUTIONS * N);
    thrust::device_vector<double>::iterator min_cost_iter;
    
    // Read from file
    for (int i = 0; i < h_points.size(); i++) {
        std::cin >> p.x;
        std::cin >> p.y;
        h_points[i] = p;
    }

    // Copy points vector to device
    thrust::device_vector<point> d_points(h_points);

    // Precompute distances in GPU
    dim3 threads_dist(32, 32, 1);
    dim3 grid_dist(ceil(N/threads_dist.x), ceil(N/threads_dist.y), 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, NULL);

    distKernel<<<grid_dist, threads_dist>>>(
        thrust::raw_pointer_cast(d_points.data()),
        thrust::raw_pointer_cast(d_distances.data()),
        N
    );

    // Each thread will generate a random solution
    solKernel<<<ceil(SOLUTIONS/1024), 1024>>>(
        thrust::raw_pointer_cast(d_distances.data()),
        thrust::raw_pointer_cast(d_costs.data()),
        thrust::raw_pointer_cast(d_solutions.data()),
        N
    );

    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);

    // Find smallest cost for best solution
    min_cost_iter = thrust::min_element(d_costs.begin(), d_costs.end());
    min_cost_pos = min_cost_iter - d_costs.begin();
    min_cost = *min_cost_iter;

    std::cerr << msecTotal << " ms" << std::endl;

    std::cout << std::fixed << std::setprecision(5);
    std::cout << min_cost << " 1" << std::endl;
    for (int i = min_cost_pos * N; i < (min_cost_pos * N) + N; i++) {
        std::cout << d_solutions[i] << " ";
    }
    std::cout << "\n";
}