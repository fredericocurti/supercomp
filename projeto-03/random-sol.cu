#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <iomanip>
#include <cmath>
#include <chrono>
#include <iostream>
#include <stdio.h>

typedef struct {
    double x;
    double y;
} point;

__device__ double dist(point p1, point p2) {
    return sqrt(pow(p1.x - p2.x, 2) + pow(p1.y - p2.y, 2));
}

__device__ void print_vec_int(int *v, int N) {
    for (int i = 0; i < N; i++) {
        printf("%d ", v[i]);
    }
    printf("\n");
}

__device__ void swap(int *a, int *b) {
    int temp = *a;
    *a = *b;
    *b = temp;
    return;
}

__global__ void distKernel(point *points, double *d_distances, int N) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N || j >= N) return;
    d_distances[i * N + j] = dist(points[i], points[j]);
}

__global__ void solKernel(double *d_distances, double *d_costs, int *d_solutions, int N) {
    hiprandState st;
    int ri; // random index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("%d\n", i);
    int cost = 0;
    hiprand_init(0, i, 0, &st);

    
    // Fill solution with sequential possible N's
    for (int j = 0; j < N; j++) {
        d_solutions[i * N + j] = j;
    }

    for (int j = 1; j < N; j++) {
        ri = (int) ((N - j) * hiprand_uniform(&st) + j); // (max - min) * hiprand + min
        
        // if (i == 0) {
        //     printf("--------  RANDOM INDEX: %d -------- \n", ri);
        //     printf("before: swapping index %d(%d) with %d(%d)\n", i * N + j, d_solutions[i * N + j],  i * N + ri, d_solutions[i * N + ri]);
        // }

        swap(&d_solutions[i * N + j], &d_solutions[i * N + ri]);

        // if (i == 0) {
        //     printf("after: index %d(%d) with %d(%d)\n", i * N + j, d_solutions[i * N + j],  i * N + ri, d_solutions[i * N + ri]);
        // }
        
        cost += d_distances[d_solutions[i * N + (j - 1)] * N + d_solutions[i * N + j]];
    }

    // Add cost between first and last
    cost += d_distances[d_solutions[i * N] * N + d_solutions[i * N + (N - 1)]];

    d_costs[i] = cost;

    // if (i == 100) {
    //     print_vec_int(d_solutions, N);
    //     printf("cost: %f\n", d_costs[i]);
    // }
}

int main() {
    // Variable declaration
    int N, min_cost_pos;
    // int SOLUTIONS = 100000;
    int SOLUTIONS = 1024;
    double min_cost;
    point p;
    std::cin >> N;
    thrust::host_vector<point> h_points(N);
    thrust::device_vector<double> d_distances(N * N);
    thrust::device_vector<double> d_costs(SOLUTIONS);
    thrust::device_vector<int> d_solutions(SOLUTIONS * N);
    thrust::device_vector<double>::iterator min_cost_iter;
    // best_cost = numeric_limits<double>::max();

    // Read from file
    for (int i = 0; i < h_points.size(); i++) {
        std::cin >> p.x;
        std::cin >> p.y;
        h_points[i] = p;
    }

    // Copy points vector to device
    thrust::device_vector<point> d_points(h_points);

    // Precompute distances in GPU
    dim3 threads_dist(32, 32, 1);
    dim3 grid_dist(ceil(N/threads_dist.x), ceil(N/threads_dist.y), 1);

    distKernel<<<grid_dist, threads_dist>>>(
        thrust::raw_pointer_cast(d_points.data()),
        thrust::raw_pointer_cast(d_distances.data()),
        N
    );

    // Each thread will generate a random solution
    solKernel<<<ceil(SOLUTIONS/1024), 1024>>>(
        thrust::raw_pointer_cast(d_distances.data()),
        thrust::raw_pointer_cast(d_costs.data()),
        thrust::raw_pointer_cast(d_solutions.data()),
        N
    );

    // Find smallest cost for best solution
    min_cost_iter = thrust::min_element(d_costs.begin(), d_costs.end());
    min_cost_pos = min_cost_iter - d_costs.begin();
    min_cost = *min_cost_iter;

    std::cout << "min cost: " << min_cost << " at index: " << min_cost_pos << std::endl;

    std::cout << "best solution: [ ";
    for (int i = min_cost_pos * N; i < (min_cost_pos * N) + N; i++) {
        std::cout << d_solutions[i] << " ";
    }
    std::cout << "]\n";

    std::cout << std::endl;

    // d = chrono::duration_cast<chrono::microseconds>(end - start).count();
    // std::cerr << d << " μs" << endl;
    // std::cout << std::fixed;
    // std::cout << setprecision(5);
    // std::cout << path_dist(best_sol, points) << " 0" << std::endl;

    // std::cout << setprecision(0);
    // for (int i = 0; i < best_sol.size(); i++) {
    //     std::cout << best_sol[i] << " ";
    // }
    // std::cout << endl;
}